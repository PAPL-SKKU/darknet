#include <boost/date_time/posix_time/posix_time.hpp>

#include <fstream>  // NOLINT(readability/streams)
#include <iostream>  // NOLINT(readability/streams)
#include <map>
#include <set>
#include <sstream>
#include <string>
#include <utility>  // pair
#include <vector>

#include "timer.hpp"

Timer::Timer()
    : initted_(false),
      running_(false),
      has_run_at_least_once_(false) {
  Init();
}

Timer::~Timer() {
#ifndef CPU_ONLY
    CUDA_CHECK(hipEventDestroy(start_gpu_));
    CUDA_CHECK(hipEventDestroy(stop_gpu_));
#else
    NO_GPU;
#endif
}

void Timer::Start() {
  if (!running()) {
#ifndef CPU_ONLY
      CUDA_CHECK(hipEventRecord(start_gpu_, 0));
#else
      NO_GPU;
#endif
    running_ = true;
    has_run_at_least_once_ = true;
  }
}

void Timer::Stop() {
  if (running()) {
#ifndef CPU_ONLY
      CUDA_CHECK(hipEventRecord(stop_gpu_, 0));
      CUDA_CHECK(hipEventSynchronize(stop_gpu_));
#else
      NO_GPU;
#endif
    running_ = false;
  }
}


float Timer::MicroSeconds() {
  if (!has_run_at_least_once()) {
    return 0;
  }
  if (running()) {
    Stop();
  }
#ifndef CPU_ONLY
    CUDA_CHECK(hipEventElapsedTime(&elapsed_milliseconds_, start_gpu_,
                                    stop_gpu_));
    // Cuda only measure milliseconds
    elapsed_microseconds_ = elapsed_milliseconds_ * 1000;
#else
      NO_GPU;
#endif
  return elapsed_microseconds_;
}

float Timer::MilliSeconds() {
  if (!has_run_at_least_once()) {
    return 0;
  }
  if (running()) {
    Stop();
  }
#ifndef CPU_ONLY
    CUDA_CHECK(hipEventElapsedTime(&elapsed_milliseconds_, start_gpu_,
                                    stop_gpu_));
#else
      NO_GPU;
#endif
  return elapsed_milliseconds_;
}

float Timer::Seconds() {
  return MilliSeconds() / 1000.;
}

void Timer::Init() {
  if (!initted()) {
#ifndef CPU_ONLY
      CUDA_CHECK(hipEventCreate(&start_gpu_));
      CUDA_CHECK(hipEventCreate(&stop_gpu_));
#else
      NO_GPU;
#endif
    initted_ = true;
  }
}
